#include "hip/hip_runtime.h"

//CUDA Library
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>

//C++ Library
#include <stdio.h>
#include <cstdio>
#include <iostream>
#include <fstream>

//Personal Library
#include "headers.hpp"
#include "smartptr.hpp"
#include "storage.h"

//OPENCV Library
#include <opencv2/core/core.hpp>

using namespace std;
using namespace cv;

//MACRO
#define SIZEOF(width, height, typevariable) sizeof(typevariable)*(width*height)
#define LOC(width, height, widthOfMat) (height*widthOfMat) + width
#define POW(width, height) width * height

//DEFINITION
const int SIZE = 2<<27;

//Prototype Function Declaration
template <unsigned int convWidth, unsigned  int widthofmat> __global__ void Map_Gather(int*, int*, int*);
__host__ void map_gather();
void curand_function();

class image_config{
public:
	image_config() = delete;
	void read_image(string fileName = "");
private:

};

class deepLearning{
public:
	void display(int width, int height, int* ptr);
private:
	int* dMem = nullptr;
	int* OdMem = nullptr;
	int* OhMem = nullptr;
	int* ConvdMem = nullptr;
	int* ConvhMem = nullptr;
	int* hMem = nullptr;

	//TEST PARM
	int print;

	const int imageSize = 10;
	const int convSize = imageSize - 2;
};


int main(int argc, char* argv[]) {

	try
	{
		map_gather();
//		curand_function();

		cout << "PROGRAM FINISHED" << endl;
	}
	catch (const Exception& excep) {
		cerr << "Error found at line " << excep.Line << " with error code " << excep.type << " and error type " << excep.type << endl;;
		return -1;
	}

	cin.get();
}

template <unsigned int convWidth, unsigned  int widthofmat> __global__ void Map_Gather(int* input, int* output, int* convlayer) {

	int loResult = 0;

	const int ix = threadIdx.x;
	const int iy = threadIdx.y;

	for (int i_y = 0; i_y < 3; ++i_y) {

		for (int i_x = 0; i_x < 3; ++i_x) {

			loResult += convlayer[LOC(i_x, i_y, 3)] * input[LOC((ix + i_x), (iy + i_y), widthofmat)];

		}

	}

	output[LOC(ix, iy, convWidth)] = loResult;

}

__host__ void map_gather() {

	cout << "RUNNING 'Map and Gather Function'" << endl;

	const int imageSize = 10;
	const int convSize = imageSize - 2;

	int* dMem = nullptr;
	int* OdMem = nullptr;
	int* OhMem = nullptr;
	int* ConvdMem = nullptr;
	int* ConvhMem = nullptr;
	int* hMem = nullptr;

	cuda(hipMalloc(&dMem, SIZEOF(imageSize, imageSize, int)));
	cuda(hipMalloc(&OdMem, SIZEOF(convSize, convSize, int)));
	cuda(hipHostAlloc(&OhMem, SIZEOF(convSize, convSize, int), hipHostMallocDefault));
	cuda(hipMalloc(&ConvdMem, SIZEOF(3, 3, int)));
	cuda(hipHostAlloc(&hMem, SIZEOF(imageSize, imageSize, int), hipHostMallocDefault));
	cuda(hipHostAlloc(&ConvhMem, SIZEOF(3, 3, int), hipHostMallocDefault));

	for (int i = 0; i < POW(imageSize, imageSize); ++i) {
		hMem[i] = 0;
	}

	hMem[LOC(1, 1, imageSize)] = 1;
	hMem[LOC(1, 2, imageSize)] = 1;
	hMem[LOC(1, 3, imageSize)] = 1;

	hMem[LOC(5, 1, imageSize)] = 1;
	hMem[LOC(4, 2, imageSize)] = 1;
	hMem[LOC(3, 3, imageSize)] = 1;

	ConvhMem[0] = 0;
	ConvhMem[1] = 0;
	ConvhMem[2] = 9;
	ConvhMem[3] = 0;
	ConvhMem[4] = 9;
	ConvhMem[5] = 0;
	ConvhMem[6] = 9;
	ConvhMem[7] = 0;
	ConvhMem[8] = 0;

	cout << "IMAGE LAYER" << endl;
//	display(imageSize, imageSize, hMem);

	cout << "CONV LAYER" << endl;
//	display(3, 3, ConvhMem);

	cuda(hipMemcpy(dMem, hMem, SIZEOF(imageSize, imageSize, int), hipMemcpyHostToDevice));
	cuda(hipMemcpy(ConvdMem, ConvhMem, SIZEOF(3, 3, int), hipMemcpyHostToDevice));

	dim3 threads(convSize, convSize);

	Map_Gather<convSize, imageSize><<< 1, threads>>>(dMem, OdMem, ConvdMem);

	cuda(hipMemcpy(OhMem, OdMem, SIZEOF(convSize, convSize, int), hipMemcpyDeviceToHost));

	cout << "ACTIVATION LAYER 1" << endl;

//	display(convSize, convSize, OhMem);

	cuda(hipFree(dMem));
	cuda(hipFree(ConvdMem));
	cuda(hipFree(OdMem));
	cuda(hipHostFree(OhMem));
	cuda(hipHostFree(hMem));
	cuda(hipHostFree(ConvhMem));

	dMem = nullptr;
	OdMem = nullptr;
	OhMem = nullptr;
	ConvdMem = nullptr;
	ConvhMem = nullptr;
	hMem = nullptr;

}

void curand_function() {

	//Declare CURAND Generator
	hiprandGenerator_t generator;

	//Create CURAND Generator
	crand(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));

	//Set CURAND Seed
	crand(hiprandSetPseudoRandomGeneratorSeed(generator, 8888LL));

	//Declare GPU MEM
	unsigned int* memNumber = nullptr;

	//Declare CPU MEM
	ptr::uPtr<unsigned int> cpuMem(new unsigned int[SIZE]);

	cuda(hipMalloc(&memNumber, sizeof(int) * SIZE));

	crand(hiprandGenerate(generator, memNumber, SIZE));

	cuda(hipMemcpy(*cpuMem, memNumber, sizeof(unsigned int)* SIZE, hipMemcpyDeviceToHost));

	storage st(cpuMem, SIZE);

	st.print(SIZE);

	//FREE MEMORY
	crand(hiprandDestroyGenerator(generator));
	cuda(hipFree(memNumber));

	//Initialize To NULL
	generator = nullptr;
	memNumber = nullptr;
}

void image_config::read_image(string fileName = ""){
	Mat image;
}

void deepLearning::display(int width, int height, int* ptr) {
	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			cout << ptr[LOC(x, y, width)] << ",";
		}

		cout << endl;
	}
}
